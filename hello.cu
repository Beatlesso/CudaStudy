
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int block_id = blockIdx.x;
    const int thread_id = threadIdx.x;
    printf("%d %d hello from gpu!\n", block_id, thread_id);
}

int main(void)
{
    hello_from_gpu<<<2, 4>>>();
    hipDeviceReset();
    hipDeviceSynchronize();
    printf("end\n");
    return 0;
}